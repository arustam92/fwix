#include "hip/hip_runtime.h"
#include <complex_vector.h>
#include <hip/hip_complex.h>

__global__ void add(hipFloatComplex* vec1, hipFloatComplex* vec2, int N) {

  int i0 = threadIdx.x + blockDim.x*blockIdx.x;
  int j = blockDim.x * gridDim.x;

  for (int i=i0; i < N; i += j)
    vec1[i] = hipCaddf(vec1[i], vec2[i]);
};
void launch_add(complex_vector* vec1, complex_vector* vec2, dim3 grid, dim3 block) {
  add<<<grid, block>>>(vec1->mat, vec2->mat, vec1->nelem);
};

