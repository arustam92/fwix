#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <iostream>

__device__ void CB_ortho(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr) {
  int* n = (int*)callerInfo;
  float norm_factor = sqrtf(1.f/float((n[0] * n[1])));
  ((hipfftComplex*)dataOut)[offset] = hipCmulf(element, make_hipComplex(norm_factor, 0.0f));
}
__device__ hipfftCallbackStoreC d_storeCallbackPtr = CB_ortho;

hipfftCallbackStoreC get_host_callback_ptr() {
  hipfftCallbackStoreC h_storeCallbackPtr;
  hipMemcpyFromSymbol(&h_storeCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_storeCallbackPtr));
  return h_storeCallbackPtr;
}